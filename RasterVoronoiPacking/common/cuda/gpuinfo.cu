#include "hip/hip_runtime.h"
#include "gpuinfo.h"

#include <iostream> // to output to the console
#include <hip/hip_runtime.h> // to get memory on the device
#include <hip/hip_runtime.h> // to get device count

#define BLOCK_SIZE 16
#define REDUCTION_BLOCK_SIZE 256
#define EPS 0.000001

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		int a;
		std::cin >> a;
		if (abort) exit(code);
	}
}

namespace CUDAPACKING {

	// Problem pointers
	CudaRasterNoFitPolygon **h_dpointerdpointers, **h_hpointerdpointers, **d_dpointerdpointer;
	int *d_itemTypeMap;
	float *d_overlapmap;
	// Solution pointers
	int *d_posx, *d_posy, *d_angles;
	float *d_weights;

	bool getTotalMemory(int &gpuDeviceCount, size_t &free, size_t &total) {
		int deviceCount, device;
		gpuDeviceCount = 0;
		struct hipDeviceProp_t properties;
		hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
		if (cudaResultCode != hipSuccess)
			deviceCount = 0;
		/* machines with no GPUs can still report one emulation device */
		for (device = 0; device < deviceCount; ++device) {
			hipGetDeviceProperties(&properties, device);
			if (properties.major != 9999) /* 9999 means emulation only */
				++gpuDeviceCount;
		}
		//printf("%d GPU CUDA device(s) found\n", gpuDeviceCount);

		/* don't just return the number of gpus, because other runtime cuda
		errors can also yield non-zero return values */
		if (gpuDeviceCount > 0) {
			hipMemGetInfo(&free, &total);
			//std::cout << "free memory: " << free / 1024 / 1024 << "mb, total memory: " << total / 1024 / 1024 << "mb" << std::endl;
			return true; /* success */
		}
		else return false; /* failure */
	}

	void allocItemTypes(int numItems) {
		gpuErrchk(hipMalloc((void**)&d_itemTypeMap, numItems*sizeof(int)));
	}

	void alloDevicecSolutionPointers(int numItems) {
		gpuErrchk(hipMalloc((void**)&d_posx, numItems*sizeof(int)));
		gpuErrchk(hipMalloc((void**)&d_posy, numItems*sizeof(int)));
		gpuErrchk(hipMalloc((void**)&d_angles, numItems*sizeof(int)));
		gpuErrchk(hipMalloc((void**)&d_weights, numItems*sizeof(float)));
	}

	void setItemType(int itemId, int typeId) {
		gpuErrchk(hipMemcpy(d_itemTypeMap + itemId, &typeId, sizeof(int), hipMemcpyHostToDevice));
	}

	void allocHostNfpPointers(int numItems, int numOrientations) {
		// Nofit polygon values
		h_dpointerdpointers = (CudaRasterNoFitPolygon**)malloc(numItems*numOrientations*sizeof(CudaRasterNoFitPolygon*));
		h_hpointerdpointers = (CudaRasterNoFitPolygon**)malloc(numItems*numOrientations*sizeof(CudaRasterNoFitPolygon*));
		for (int l = 0; l < numItems*numOrientations; l++)
			h_hpointerdpointers[l] = (CudaRasterNoFitPolygon*)malloc(numItems*numOrientations*sizeof(CudaRasterNoFitPolygon));
	}

	void allocSingleDeviceNfpMatrix(int staticId, int orbitingId, int *matrix, int width, int height, int originx, int originy) {
		h_hpointerdpointers[staticId][orbitingId].setOrigin(cuPoint(originx, originy));
		h_hpointerdpointers[staticId][orbitingId].setWidth(width);
		h_hpointerdpointers[staticId][orbitingId].setHeight(height);
		gpuErrchk(hipMalloc((void**)&h_hpointerdpointers[staticId][orbitingId].matrix, width*height*sizeof(int)));
		gpuErrchk(hipMemcpy(h_hpointerdpointers[staticId][orbitingId].matrix, matrix, width*height*sizeof(int), hipMemcpyHostToDevice));
	}

	void allocDeviceNfpPointers(int numItems, int numOrientations) {
		for (int l = 0; l < numItems*numOrientations; l++) {
			gpuErrchk(hipMalloc((void**)&h_dpointerdpointers[l], numItems*numOrientations*sizeof(CudaRasterNoFitPolygon)));
			gpuErrchk(hipMemcpy(h_dpointerdpointers[l], h_hpointerdpointers[l], numItems*numOrientations*sizeof(CudaRasterNoFitPolygon), hipMemcpyHostToDevice));
		}
		gpuErrchk(hipMalloc((void**)&d_dpointerdpointer, numItems*numOrientations*sizeof(CudaRasterNoFitPolygon*)));
		gpuErrchk(hipMemcpy(d_dpointerdpointer, h_dpointerdpointers, numItems*numOrientations*sizeof(CudaRasterNoFitPolygon*), hipMemcpyHostToDevice));
	}

	void allocDeviceMaxIfp(size_t memSize) {
		gpuErrchk(hipMalloc((void**)&d_overlapmap, memSize));
	}

	// GPU displaced sum of two matrix. TODO: Store nfp widths, heights and origins in shared memory.
	__global__ static void DisplacedSumKernel(float *d_overlapmap, int omwidth, int omheight, int overlapmapx, int overlapmapy, int nAngles, CudaRasterNoFitPolygon **nfpSet, int nfpcount, int *itemType, int itemId, int itemAngle, int *posx, int *posy, int *angles)
	{
		const int tidi = blockDim.x * blockIdx.x + threadIdx.x;
		const int tidj = blockDim.y * blockIdx.y + threadIdx.y;
		const int orbitingId = itemType[itemId]*nAngles + itemAngle;

		int nfpCoordx, nfpCoordy;
		float tempVal = 0;
		if (tidi < omwidth && tidj < omheight) {
			for (int k = 0; k < nfpcount; k++) {
				if (k == itemId) continue;
				int staticId = itemType[k] * nAngles + angles[k];
				nfpCoordx = tidi - overlapmapx - posx[k] + nfpSet[staticId][orbitingId].origin.x;
				nfpCoordy = tidj - overlapmapy - posy[k] + nfpSet[staticId][orbitingId].origin.y;
				if (nfpCoordx >= 0 && nfpCoordx < nfpSet[staticId][orbitingId].m_width && nfpCoordy >= 0 && nfpCoordy < nfpSet[staticId][orbitingId].m_height)
					tempVal += (float)nfpSet[staticId][orbitingId].matrix[nfpCoordy*nfpSet[staticId][orbitingId].m_width + nfpCoordx];
			}
			d_overlapmap[tidj*omwidth + tidi] = tempVal;
		}
	}

	// GPU displaced sum of two matrix with weights. TODO: Store nfp widths, heights and origins in shared memory.
	__global__ static void DisplacedWeightedSumKernel(float *d_overlapmap, int omwidth, int omheight, int overlapmapx, int overlapmapy, int nAngles, CudaRasterNoFitPolygon **nfpSet, int nfpcount, int *itemType, int itemId, int itemAngle, int *posx, int *posy, int *angles, float *weights)
	{
		const int tidi = blockDim.x * blockIdx.x + threadIdx.x;
		const int tidj = blockDim.y * blockIdx.y + threadIdx.y;
		const int orbitingId = itemType[itemId] * nAngles + itemAngle;

		int nfpCoordx, nfpCoordy;
		float tempVal = 0;
		if (tidi < omwidth && tidj < omheight) {
			for (int k = 0; k < nfpcount; k++) {
				if (k == itemId) continue;
				int staticId = itemType[k] * nAngles + angles[k];
				nfpCoordx = tidi - overlapmapx - posx[k] + nfpSet[staticId][orbitingId].origin.x;
				nfpCoordy = tidj - overlapmapy - posy[k] + nfpSet[staticId][orbitingId].origin.y;
				if (nfpCoordx >= 0 && nfpCoordx < nfpSet[staticId][orbitingId].m_width && nfpCoordy >= 0 && nfpCoordy < nfpSet[staticId][orbitingId].m_height)
					tempVal += weights[k]*(float)nfpSet[staticId][orbitingId].matrix[nfpCoordy*nfpSet[staticId][orbitingId].m_width + nfpCoordx];
			}
			d_overlapmap[tidj*omwidth + tidi] = tempVal;
		}
	}

	__global__ void FindMinimumPositionKernel(float *g_idata, float *g_odata, int *g_opos, int n, bool first) {
		extern __shared__ float smem[];

		float *sdata = smem;
		int *sdatapos = (int*)&sdata[REDUCTION_BLOCK_SIZE];

		// each thread loads one element from global to shared mem
		unsigned int tid = threadIdx.x;
		unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
		if (i < n) {
			sdata[tid] = g_idata[i];
			if (first) sdatapos[tid] = i; else sdatapos[tid] = g_opos[i];
		}
		else {
			sdata[tid] = g_idata[n - 1];
			if (first) sdatapos[tid] = n - 1; else sdatapos[tid] = g_opos[n - 1];
		}
		__syncthreads();

		// do reduction in shared mem
		for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
			if (tid < s) {
				if (abs(sdata[tid] - sdata[tid + s]) < FLT_EPSILON) {
					if(sdatapos[tid + s] < sdatapos[tid]) {
						sdata[tid] = sdata[tid + s];
						sdatapos[tid] = sdatapos[tid + s];
					}
				}
				else {
					if (sdata[tid + s] < sdata[tid]) {
						sdata[tid] = sdata[tid + s];
						sdatapos[tid] = sdatapos[tid + s];
					}
				}
			}
			__syncthreads();
		}

		// write result for this block to global mem
		if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; g_opos[blockIdx.x] = sdatapos[0]; }
	}

	// Copy position and angles to device
	void uploadSolutionParameters(int *posx, int *posy, int *angles, int nItems) {
		hipMemcpy(d_posx, posx, nItems*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_posy, posy, nItems*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_angles, angles, nItems*sizeof(int), hipMemcpyHostToDevice);
	}

	// Copy position, angles and weights to device
	void uploadSolutionParameters(int *posx, int *posy, int *angles, float *weights, int nItems) {
		hipMemcpy(d_posx, posx, nItems*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_posy, posy, nItems*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_angles, angles, nItems*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_weights, weights, nItems*sizeof(float), hipMemcpyHostToDevice);
	}

	// Create overlap map on device
	void detOverlapMapOnDevice(int curItem, int curItemAngle, int nItems, int numAngles, int overlapmap_width, int overlapmap_height, int overlapmapx, int overlapmapy, bool useGlsWeights) {
		// Execute Kernel to determine the map
		dim3 blocks(1, 1, 1);
		dim3 threadsperblock(BLOCK_SIZE, BLOCK_SIZE, 1);
		blocks.x = ((overlapmap_width / BLOCK_SIZE) + (((overlapmap_width) % BLOCK_SIZE) == 0 ? 0 : 1));
		blocks.y = ((overlapmap_height / BLOCK_SIZE) + (((overlapmap_height) % BLOCK_SIZE) == 0 ? 0 : 1));
		if (!useGlsWeights) DisplacedSumKernel << <blocks, threadsperblock >> >(d_overlapmap, overlapmap_width, overlapmap_height, overlapmapx, overlapmapy, numAngles, d_dpointerdpointer, nItems, d_itemTypeMap, curItem, curItemAngle, d_posx, d_posy, d_angles);
		else DisplacedWeightedSumKernel << <blocks, threadsperblock >> >(d_overlapmap, overlapmap_width, overlapmap_height, overlapmapx, overlapmapy, numAngles, d_dpointerdpointer, nItems, d_itemTypeMap, curItem, curItemAngle, d_posx, d_posy, d_angles, d_weights);
	}

	// Returns a pointer to an overlap map on host
	float *getcuOverlapMap(int curItem, int curItemAngle, int nItems, int numAngles, int overlapmap_width, int overlapmap_height, int overlapmapx, int overlapmapy, int *posx, int *posy, int *angles, float *weights, bool useGlsWeights) {
		// Upload solution parameters to GPU
		if (useGlsWeights) uploadSolutionParameters(posx, posy, angles, weights, nItems);
		else uploadSolutionParameters(posx, posy, angles, nItems);

		// Determine overlap map on GPU
		detOverlapMapOnDevice(curItem, curItemAngle, nItems, numAngles, overlapmap_width, overlapmap_height, overlapmapx, overlapmapy, useGlsWeights);

		// Copy overlap map result to host
		float *h_overlapmap = (float *)malloc(overlapmap_width*overlapmap_height*sizeof(float));
		hipMemcpy(h_overlapmap, d_overlapmap, overlapmap_width * overlapmap_height * sizeof(float), hipMemcpyDeviceToHost);
		return h_overlapmap;
	}

	// Create overlap map on device
	void detOverlapMapOnDevice(float **d_valueVec, int **d_PosVec, float *d_map, int width, int height) {
		// Execute Kernel to determine the minimum position
		dim3 blocks2(1, 1, 1);
		dim3 threadsperblock2(REDUCTION_BLOCK_SIZE, 1, 1);
		blocks2.x = ((width*height) / REDUCTION_BLOCK_SIZE) + (((width*height) % REDUCTION_BLOCK_SIZE) == 0 ? 0 : 1);
		gpuErrchk(hipMalloc((void**) &(*d_valueVec), blocks2.x*sizeof(float)));
		gpuErrchk(hipMalloc((void**) &(*d_PosVec), blocks2.x*sizeof(int)));
		FindMinimumPositionKernel << <blocks2, threadsperblock2, REDUCTION_BLOCK_SIZE*sizeof(float)+REDUCTION_BLOCK_SIZE*sizeof(int) >> >(d_map, *d_valueVec, *d_PosVec, width*height, true);
		int reducedsize = blocks2.x;
		while (reducedsize > 1) {
			blocks2.x = ((reducedsize / REDUCTION_BLOCK_SIZE) + (((reducedsize) % REDUCTION_BLOCK_SIZE) == 0 ? 0 : 1));
			FindMinimumPositionKernel << <blocks2, threadsperblock2, REDUCTION_BLOCK_SIZE*sizeof(float)+REDUCTION_BLOCK_SIZE*sizeof(int) >> >(*d_valueVec, *d_valueVec, *d_PosVec, reducedsize, false);
			reducedsize = blocks2.x;
		}
	}

	// Returns minimum overlap position and value using GPU. TODO: Change placement in gpu memory.
	float getcuMinimumOverlap(int curItem, int curItemAngle, int nItems, int numAngles, int overlapmap_width, int overlapmap_height, int overlapmapx, int overlapmapy, int *posx, int *posy, int *angles, float *weights, int &minX, int &minY, bool useGlsWeights) {
		// Upload solution parameters to GPU
		if (useGlsWeights) uploadSolutionParameters(posx, posy, angles, weights, nItems);
		else uploadSolutionParameters(posx, posy, angles, nItems);

		// Determine overlap map on GPU
		detOverlapMapOnDevice(curItem, curItemAngle, nItems, numAngles, overlapmap_width, overlapmap_height, overlapmapx, overlapmapy, useGlsWeights);

		// Execute Kernel to determine the minimum position
		float *d_temp_output; int *d_temp_pos;
		detOverlapMapOnDevice(&d_temp_output, &d_temp_pos, d_overlapmap, overlapmap_width, overlapmap_height);

		// Copy result to host
		float minVal; int linearPosition; 
		hipMemcpy(&minVal, d_temp_output, sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(&linearPosition, d_temp_pos, sizeof(int), hipMemcpyDeviceToHost);
		minX = - overlapmapx + linearPosition % overlapmap_width; minY = - overlapmapy + linearPosition / overlapmap_width;

		// Free GPU temporary pointers
		gpuErrchk(hipFree(d_temp_output));
		gpuErrchk(hipFree(d_temp_pos));

		return minVal;
	}
}