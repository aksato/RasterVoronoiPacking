#include "hip/hip_runtime.h"
#include "rasteroverlapevaluatorcudafull.h"

#define THREADS_PER_BLOCK 512
typedef quint32* NfpData;
using namespace RASTERVORONOIPACKING;

__device__
int getRasterNoFitPolygonKey(int staticPieceTypeId, int staticAngleId, int orbitingPieceTypeId, int orbitingAngleId, int numAngles, int numKeys) {
	int staticKey = staticPieceTypeId * numAngles + staticAngleId;
	int orbitingKey = orbitingPieceTypeId * numAngles + orbitingAngleId;
	return staticKey + numKeys * orbitingKey;
}

__device__
int getWeight(unsigned int* weights, int itemId1, int itemId2, int numItems) {
	if (itemId1 > itemId2) return weights[itemId1 + numItems * itemId2];
	return weights[itemId2 + numItems * itemId1];
}

__global__
void cudaGetTotalOverlapMap(quint32* map, int width, int height, int referencePointX, int referencePointY, DeviceRasterNoFitPolygonSet nfps, DeviceRasterPackingSolution solution,
	int numItems, int *itemType, int itemId, int orientation, int numAngles, int numKeys, unsigned int *weights) {
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i >= width || j >= height)
		return;

	for (int k = 0; k < numItems; k++) {
	//for (int k = 18; k < 19; k++) {
		if (k == itemId) continue;
		int nfpKey = getRasterNoFitPolygonKey(itemType[k], solution.d_orientations[k], itemType[itemId], orientation, numAngles, numKeys);
		NfpData nfp = nfps.d_data[nfpKey];
		int nfpWidth = nfps.d_widths[nfpKey]; int nfpHeight = nfps.d_heights[nfpKey];
		int nfpOriginX = nfps.d_originsX[nfpKey]; int nfpOriginY = nfps.d_originsY[nfpKey];
		int nfpMutiplier = nfps.d_multipliers[nfpKey];
		int relativeOriginX = referencePointX + solution.d_posX[k] - nfpMutiplier * nfpOriginX + (nfpWidth - 1) * (nfpMutiplier - 1) / 2;
		int relativeOriginY = referencePointY + solution.d_posY[k] - nfpMutiplier * nfpOriginY + (nfpHeight - 1) * (nfpMutiplier - 1) / 2;

		if (i < relativeOriginX || i > relativeOriginX + nfpWidth - 1 || j < relativeOriginY || j > relativeOriginY + nfpHeight - 1) {}
		else {
			int nfpidx = nfpMutiplier < 0 ? 
				nfpWidth * nfpHeight - 1 - j + relativeOriginY - (i - relativeOriginX) * nfpHeight :
				j - relativeOriginY + (i - relativeOriginX) * nfpHeight;
			map[i * height + j] += nfp[nfpidx] * getWeight(weights, itemId, k, numItems);
		}
	}
}

RasterTotalOverlapMapEvaluatorCudaFull::RasterTotalOverlapMapEvaluatorCudaFull(std::shared_ptr<RasterPackingProblem> _problem) :
	RasterTotalOverlapMapEvaluatorCudaGLS(_problem) {
	glsWeightsCuda = std::shared_ptr<GlsWeightSetCuda>(new GlsWeightSetCuda(_problem->count()));
	initCuda(_problem);
}

RasterTotalOverlapMapEvaluatorCudaFull::RasterTotalOverlapMapEvaluatorCudaFull(std::shared_ptr<RasterPackingProblem> _problem, std::shared_ptr<GlsWeightSetCuda> _glsWeightsCuda) : 
	RasterTotalOverlapMapEvaluatorCudaGLS(_problem, _glsWeightsCuda), glsWeightsCuda(_glsWeightsCuda) {
	initCuda(_problem);
}

RasterTotalOverlapMapEvaluatorCudaFull::~RasterTotalOverlapMapEvaluatorCudaFull() {
	hipFree(d_nfps.d_data);
	hipFree(d_nfps.d_widths);
	hipFree(d_nfps.d_heights);
	hipFree(d_nfps.d_originsX);
	hipFree(d_nfps.d_originsY);
	hipFree(d_nfps.d_multipliers);
	hipFree(d_itemId2ItemTypeMap);
}

void RasterTotalOverlapMapEvaluatorCudaFull::initCuda(std::shared_ptr<RasterPackingProblem> _problem) {
	numKeys = _problem->getNfps()->getSize();
	numAngles = problem->getNfps()->getNumAngles();

	NfpData* data;
	int* widths, * heights;
	int* originsX, * originsY, * multipliers;
	data = new NfpData[numKeys * numKeys];
	widths = new int[numKeys * numKeys]; heights = new int[numKeys * numKeys];
	originsX = new int[numKeys * numKeys]; originsY = new int[numKeys * numKeys];
	multipliers = new int[numKeys * numKeys];

	for (int itemTypeId1 = 0; itemTypeId1 < problem->getItemTypeCount(); itemTypeId1++) {
		for (uint angle1 = 0; angle1 < (*problem->getItemByType(itemTypeId1))->getAngleCount(); angle1++) {
			for (int itemTypeId2 = 0; itemTypeId2 < problem->getItemTypeCount(); itemTypeId2++) {
				for (uint angle2 = 0; angle2 < (*problem->getItemByType(itemTypeId2))->getAngleCount(); angle2++) {
					int staticKey = itemTypeId1 * numAngles + angle1;
					int orbitingKey = itemTypeId2 * numAngles + angle2;
					std::shared_ptr<RasterNoFitPolygon> curNfp = problem->getNfps()->getRasterNoFitPolygon(itemTypeId1, angle1, itemTypeId2, angle2);
					data[staticKey + numKeys * orbitingKey] = curNfp->getMatrix();
					widths[staticKey + numKeys * orbitingKey] = curNfp->width();
					heights[staticKey + numKeys * orbitingKey] = curNfp->height();
					originsX[staticKey + numKeys * orbitingKey] = curNfp->getOrigin().x();
					originsY[staticKey + numKeys * orbitingKey] = curNfp->getOrigin().y();
					multipliers[staticKey + numKeys * orbitingKey] = curNfp->getFlipMultiplier();
				}
			}
		}
	}

	//allocation
	hipMalloc((void**)&d_nfps.d_data, (numKeys * numKeys) * sizeof(NfpData));
	hipMalloc((void**)&d_nfps.d_widths, (numKeys * numKeys) * sizeof(int));
	hipMalloc((void**)&d_nfps.d_heights, (numKeys * numKeys) * sizeof(int));
	hipMalloc((void**)&d_nfps.d_originsX, (numKeys * numKeys) * sizeof(int));
	hipMalloc((void**)&d_nfps.d_originsY, (numKeys * numKeys) * sizeof(int));
	hipMalloc((void**)&d_nfps.d_multipliers, (numKeys * numKeys) * sizeof(int));

	//copying from host to device
	hipMemcpy(d_nfps.d_data, data, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nfps.d_widths, widths, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nfps.d_heights, heights, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nfps.d_originsX, originsX, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nfps.d_originsY, originsY, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nfps.d_multipliers, multipliers, (numKeys * numKeys) * sizeof(int), hipMemcpyHostToDevice);


	int* itemId2ItemTypeMap = new int[_problem->count()];
	for (int k = 0; k < _problem->count(); k++)
		itemId2ItemTypeMap[k] = _problem->getItemType(k);
	hipMalloc((void**)&d_itemId2ItemTypeMap, _problem->count() * sizeof(int));
	hipMemcpy(d_itemId2ItemTypeMap, itemId2ItemTypeMap, _problem->count() * sizeof(int), hipMemcpyHostToDevice);

	delete[] data;
	delete[] widths;
	delete[] heights;
	delete[] originsX;
	delete[] originsY;
	delete[] multipliers;
	delete[] itemId2ItemTypeMap;
}

void RasterTotalOverlapMapEvaluatorCudaFull::updateWeights(RasterPackingSolution &solution) {
	QVector<WeightIncrement> solutionOverlapValues;

	// Determine pair overlap values
	for (int itemId1 = 0; itemId1 < problem->count(); itemId1++)
		for (int itemId2 = 0; itemId2 < problem->count(); itemId2++) {
			if (itemId1 == itemId2) continue;
			quint32 curOValue = problem->getDistanceValue(itemId1, solution.getPosition(itemId1), solution.getOrientation(itemId1),
				itemId2, solution.getPosition(itemId2), solution.getOrientation(itemId2));
			if (curOValue != 0) {
				solutionOverlapValues.append(WeightIncrement(itemId1, itemId2, 1));
			}
		}

	// Add to the current weight map
	glsWeightsCuda->updateWeights(solutionOverlapValues);
	// Update on GPU
	glsWeightsCuda->updateCudaWeights();
}

void RasterTotalOverlapMapEvaluatorCudaFull::updateWeights(RasterPackingSolution &solution, QVector<quint32> &overlaps, quint32 maxOverlap) {
	std::transform(glsWeightsCuda->begin(), glsWeightsCuda->end(), overlaps.begin(),
		glsWeightsCuda->begin(), [&maxOverlap](const quint32 &a, const quint32 &b) {return a + qRound(100.0*(qreal)b / (qreal)maxOverlap); });
	// Update on GPU
	glsWeightsCuda->updateCudaWeights();
}


//  TODO: Update cache information!
void RasterTotalOverlapMapEvaluatorCudaFull::resetWeights() {
	glsWeightsCuda->reset(problem->count());
}

// Determines the item total overlap map for a given orientation in a solution
std::shared_ptr<TotalOverlapMap> RasterTotalOverlapMapEvaluatorCudaFull::getTotalOverlapMap(int itemId, int orientation, RasterPackingSolution& solution) {
	std::shared_ptr<TotalOverlapMap> currrentPieceMap = cudamaps.getOverlapMap(itemId, orientation);
	currrentPieceMap->reset();

	DeviceRasterPackingSolution d_solution;
	int* posX = new int[problem->count()];
	int* posY = new int[problem->count()];
	int *orientations = new int[problem->count()];
	for (int k = 0; k < problem->count(); k++) {
		posX[k] = solution.getPosition(k).x();
		posY[k] = solution.getPosition(k).y();
		orientations[k] = solution.getOrientation(k);
	}
	hipMalloc((void**)&d_solution.d_posX, (problem->count()) * sizeof(int));
	hipMalloc((void**)&d_solution.d_posY, (problem->count()) * sizeof(int));
	hipMalloc((void**)&d_solution.d_orientations, (problem->count()) * sizeof(int));
	hipMemcpy(d_solution.d_posX, posX, problem->count() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_solution.d_posY, posY, problem->count() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_solution.d_orientations, orientations, problem->count() * sizeof(int), hipMemcpyHostToDevice);
	delete[] posX;
	delete[] posY;
	delete[] orientations;

	int numBlocksX = (currrentPieceMap->getWidth() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	int numBlocksY = (currrentPieceMap->getHeight() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	dim3 numBlocks(numBlocksX, numBlocksY);
	dim3 numThreads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	hipDeviceSynchronize();
	cudaGetTotalOverlapMap <<< numThreads, numBlocks >>> (
		currrentPieceMap->getData(), currrentPieceMap->getWidth(), currrentPieceMap->getHeight(), currrentPieceMap->getReferencePoint().x(), currrentPieceMap->getReferencePoint().y(),
		d_nfps, d_solution, problem->count(), d_itemId2ItemTypeMap, itemId, orientation, numAngles, numKeys, glsWeightsCuda->getCudaWeights(0));

	hipFree(d_solution.d_posX);
	hipFree(d_solution.d_posY);
	hipFree(d_solution.d_orientations);
	
	return currrentPieceMap;
}