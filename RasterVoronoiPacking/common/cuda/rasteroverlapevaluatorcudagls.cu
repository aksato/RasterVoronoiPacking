#include "cuda/rasteroverlapevaluatorcudagls.h"

using namespace RASTERVORONOIPACKING;

RasterTotalOverlapMapEvaluatorCudaGLS::RasterTotalOverlapMapEvaluatorCudaGLS(std::shared_ptr<RasterPackingProblem> _problem, bool cache) : RasterTotalOverlapMapEvaluator(_problem, false), cudamaps(_problem->count()) {
	populateMaps(cache);
}

RasterTotalOverlapMapEvaluatorCudaGLS::RasterTotalOverlapMapEvaluatorCudaGLS(std::shared_ptr<RasterPackingProblem> _problem, std::shared_ptr<GlsWeightSet> _glsWeights, bool cache) : RasterTotalOverlapMapEvaluator(_problem, _glsWeights, false), cudamaps(_problem->count()) {
	populateMaps(cache);
}

void RasterTotalOverlapMapEvaluatorCudaGLS::populateMaps(bool cache) {
	for (int itemId = 0; itemId < problem->count(); itemId++) {
		for (uint angle = 0; angle < problem->getItem(itemId)->getAngleCount(); angle++) {
			std::shared_ptr<TotalOverlapMapCuda> curMap = std::shared_ptr<TotalOverlapMapCuda>(new TotalOverlapMapCuda(problem->getIfps()->getRasterNoFitPolygon(0, 0, problem->getItemType(itemId), angle), -1));
			cudamaps.addOverlapMap(itemId, angle, curMap);
			// FIXME: Delete innerift polygons as they are used to release memomry
		}
	}
}

void RasterTotalOverlapMapEvaluatorCudaGLS::updateMapsLength(int pixelWidth) {
	int deltaPixel = problem->getContainerWidth() - pixelWidth;
	cudamaps.setShrinkVal(deltaPixel);
	for (int itemId = 0; itemId < problem->count(); itemId++)
		for (uint angle = 0; angle < problem->getItem(itemId)->getAngleCount(); angle++) {
			std::shared_ptr<TotalOverlapMap> curMap = cudamaps.getOverlapMap(itemId, angle);
			curMap->setRelativeWidth(deltaPixel);
		}
}

std::shared_ptr<TotalOverlapMap> RasterTotalOverlapMapEvaluatorCudaGLS::getTotalOverlapMap(int itemId, int orientation, RasterPackingSolution &solution) {
	std::shared_ptr<TotalOverlapMap> currrentPieceMap = cudamaps.getOverlapMap(itemId, orientation);
	currrentPieceMap->reset();
	std::shared_ptr<ItemRasterNoFitPolygonSet> curItemNfpSet = problem->getNfps()->getItemRasterNoFitPolygonSet(problem->getItemType(itemId), orientation);
	for (int i = 0; i < problem->count(); i++) {
		if (i == itemId) continue;
		currrentPieceMap->addVoronoi(i, curItemNfpSet->getRasterNoFitPolygon(problem->getItemType(i), solution.getOrientation(i)), solution.getPosition(i), getWeight(itemId, i));
	}
	return currrentPieceMap;
}

std::shared_ptr<RASTERVORONOIPACKING::TotalOverlapMap> RasterTotalOverlapMapEvaluatorCudaGLS::getOverlapMapFromDevice(std::shared_ptr<RASTERVORONOIPACKING::TotalOverlapMap> deviceMap) {
	std::shared_ptr<TotalOverlapMap> hostMap = std::shared_ptr<TotalOverlapMap>(new TotalOverlapMap(deviceMap->getRect(), deviceMap->getCuttingStockLength()));
	hipMemcpy(hostMap->getData(), deviceMap->getData(), deviceMap->getHeight() * deviceMap->getWidth() * sizeof(quint32), hipMemcpyDeviceToHost);
	return hostMap;
}