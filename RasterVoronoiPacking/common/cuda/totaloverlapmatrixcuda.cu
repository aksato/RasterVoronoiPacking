#include "hip/hip_runtime.h"
#include "cuda/totaloverlapmatrixcuda.h"
using namespace RASTERVORONOIPACKING;

#define THREADS_PER_BLOCK 512

__global__
void filloverlapmatrix(int totalLines, int lineLength, int mapInitIdx, int nfpInitIdx, int mapOffsetHeight, int nfpOffsetHeight, quint32 *map, quint32 *nfp, int multiplier)
{
	int mapidx = mapInitIdx + blockIdx.x * (lineLength + mapOffsetHeight) + blockIdx.y * blockDim.y + threadIdx.x;
	int nfpidx = multiplier * (nfpInitIdx + blockIdx.x * (lineLength + nfpOffsetHeight) + blockIdx.y * blockDim.y + threadIdx.x);
	if (threadIdx.x < lineLength)
		map[mapidx] = nfp[nfpidx];
}

TotalOverlapMatrixCuda::TotalOverlapMatrixCuda(std::shared_ptr<RasterNoFitPolygon> ifp, int _numItems, std::vector<hipStream_t> &_streams, int _cuttingStockLength) : TotalOverlapMap(ifp, _cuttingStockLength), numItems(_numItems), streams(_streams) {
	delete[] data;
	initCuda(width, height);
}

TotalOverlapMatrixCuda::TotalOverlapMatrixCuda(QRect &boundingBox, int _numItems, std::vector<hipStream_t> &_streams, int _cuttingStockLength) : TotalOverlapMap(boundingBox, _cuttingStockLength), numItems(_numItems), streams(_streams) {
	delete[] data;
	initCuda(width, height);
}

TotalOverlapMatrixCuda::TotalOverlapMatrixCuda(int width, int height, QPoint _reference, int _numItems, std::vector<hipStream_t> &_streams, int _cuttingStockLength) : TotalOverlapMap(width, height, _reference, _cuttingStockLength), numItems(_numItems), streams(_streams) {
	delete[] data;
	initCuda(width, height);
}

TotalOverlapMatrixCuda::~TotalOverlapMatrixCuda() {
	hipFree(data);
	data = nullptr;
}

void TotalOverlapMatrixCuda::initCuda(uint _width, uint _height) {
	hipMalloc((void **)&data, _width * _height * numItems * sizeof(quint32));
	hipDeviceSynchronize();
	auto error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error allocating total overlap matrix of size %f MB: %s\n", (float)(_width * _height * sizeof(quint32)) / 1024.0, hipGetErrorString(error));
		// show memory usage of GPU
		size_t free_byte, total_byte;
		auto cuda_status = hipMemGetInfo(&free_byte, &total_byte);
		if (hipSuccess != cuda_status) printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
		else {
			double free_db = (double)free_byte;
			double total_db = (double)total_byte;
			double used_db = total_db - free_db;
			printf("Memory report:: used = %.2f MB, free = %.2f MB, total = %.2f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
		}
	}
	else hipMemset(data, 0, _width * _height * numItems * sizeof(quint32));
}

void TotalOverlapMatrixCuda::setDimensions(int _width, int _height) {
	hipFree(data);
	initCuda(_width, _height);
	this->width = _width; this->height = _height;
}

void TotalOverlapMatrixCuda::reset(){
	for(int itemId = 0; itemId < numItems; itemId++)
		hipMemsetAsync(data + width * height * itemId, 0, width * height * sizeof(quint32), streams[itemId]);
}

void TotalOverlapMatrixCuda::addVoronoi(int itemId, std::shared_ptr<RasterNoFitPolygon> nfp, QPoint pos) {
	// Get intersection between innerfit and nofit polygon bounding boxes
	QPoint relativeOrigin = this->reference + pos - nfp->getFlipMultiplier() * nfp->getOrigin() + QPoint(nfp->width() - 1, nfp->height() - 1) * (nfp->getFlipMultiplier() - 1) / 2;
	int relativeBotttomLeftX = relativeOrigin.x() < 0 ? -relativeOrigin.x() : 0;
	int relativeBotttomLeftY = relativeOrigin.y() < 0 ? -relativeOrigin.y() : 0;
	int relativeTopRightX = width - relativeOrigin.x(); relativeTopRightX = relativeTopRightX < nfp->width() ? relativeTopRightX - 1 : nfp->width() - 1;
	int relativeTopRightY = height - relativeOrigin.y(); relativeTopRightY = relativeTopRightY < nfp->height() ? relativeTopRightY - 1 : nfp->height() - 1;

	// Create pointers to initial positions and calculate offsets for moving vertically
	int offsetHeight = height - (relativeTopRightY - relativeBotttomLeftY + 1);
	int nfpOffsetHeight = nfp->height() - (relativeTopRightY - relativeBotttomLeftY + 1);
	int mapInitIdx = itemId * width * height + (relativeBotttomLeftX + relativeOrigin.x())*height + relativeBotttomLeftY + relativeOrigin.y();
	int nfpInitIdx = relativeBotttomLeftY + relativeBotttomLeftX * nfp->height();

	int totalLines = relativeTopRightX - relativeBotttomLeftX + 1;
	int lineLength = relativeTopRightY - relativeBotttomLeftY + 1;
	dim3 numBlocks(totalLines, (lineLength + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
	filloverlapmatrix << < numBlocks, THREADS_PER_BLOCK, 0, streams[itemId] >> >(totalLines, lineLength, mapInitIdx, nfpInitIdx, offsetHeight, nfpOffsetHeight, data, nfp->getPixelRef(0, 0), nfp->getFlipMultiplier());
}